#include "hip/hip_runtime.h"
#include "cudaMul.cuh"
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template<typename T>
__global__  void multiplyGPU
    (
        T * Ycuda, 
        T * Acuda, 
        T * xcuda,
        int m,
        int n,
        int width,
        T spacing
    )
{
    int blkindex = blockIdx.x*blockDim.x;
    int blkindex_lookahead = blockIdx.x*blockDim.x+blockDim.x;
    int Aindex = threadIdx.x+1;
    T value =0;
    int j=0;
  
    int i = blkindex;
    for( ; i < blkindex_lookahead ; i++)
    {
        value = value + (Acuda[i])*xcuda[(j+threadIdx.x+(blockDim.x-1)*j)];
        printf("iterator value = i = %d threadIdx = %d ,value = %d , block = %d , A %d, x %d \n",i,threadIdx.x,value, blockIdx.x,*(Acuda+i),(j+threadIdx.x+(blockDim.x-1)*j));
        j++;
    }

    Ycuda[threadIdx.x+blkindex] = value;
}


template<typename T>
__global__  void init
    (
        T * Ycuda, 
        T * Acuda, 
        T * xcuda,

        int m,
        int n
     )
{

}


template<typename T>
void multiplyMatrixGpuWrapper(
        void * Ycuda, 
        void * Acuda,
        void * xcuda,

        T * Y, 
        T * A, 
        T * x,

        int m,
        int n,
        int width,
        T spacing
        ) 
{
    // dim3 block_dim(1,1,1);
    // dim3 grid_dim(1,1,1);

    std::cout<<"In the wrapper functions \n";
    int size = m*n*sizeof(T);
    std::cout<<*(A+1)<<std::endl;
    std::cout<<size<<std::endl;

    gpuErrchk(hipMalloc(&(Acuda),size));
    gpuErrchk(hipMemcpy(Acuda,A ,size, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(xcuda),size)); 
    gpuErrchk(hipMemcpy(xcuda,x , size,hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&(Ycuda),size));
    gpuErrchk(hipMemcpy(Ycuda,Y , size,hipMemcpyHostToDevice));

    // std::cout<<Acuda<<std::endl;

    multiplyGPU<<<3, 3>>>((T*)Ycuda, (T*)Acuda,(T*)xcuda, m,n ,width,spacing);
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(Y,Ycuda ,size, hipMemcpyDeviceToHost));

    hipFree(Ycuda);
    hipFree(xcuda);
    hipFree(Acuda);

}


template void multiplyMatrixGpuWrapper<int>
(
    void * Ycuda, void * Acuda, void * xcuda,
    int * Y, int * A, int * x,
    int m,int n,int width,int type
);
template void multiplyMatrixGpuWrapper<short>
(
    void * Ycuda, void * Acuda, void * xcuda,
    short * Y, short * A, short * x,
    int m,int n,int width,short type
);

template void multiplyMatrixGpuWrapper<long long>
(
    void * Ycuda, void * Acuda, void * xcuda,
    long long* Y, long long * A, long long* x,
    int m,int n,int width,long long type
);

