#include "hip/hip_runtime.h"
#include "cudaMul.cuh"
#include <iostream>

template<typename T>
__global__  void multiplyGPU
    (
        T * Ycuda, 
        T * Acuda, 
        T * xcuda,
        int m,
        int n,
        int width,
        T spacing
     )
{
  int r = blockIdx.y * blockDim.y + threadIdx.y;   
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  // check boundry conditions
  if( r < m && c < n){
    // do the multiplication for one row and col
    T value = 0;
    for(int k = 0; k < 9; k++){
      value += (*(Acuda+r * width + k)) * (*(xcuda+(k * n) + c));
    }
    // store the result
    *(Ycuda+c * n + c) = value;
    printf("%d ", *(Ycuda+r * n + c));
  }
}


template<typename T>
__global__  void init
    (
        void * Ycuda, 
        void * Acuda, 
        void * xcuda,

        T * Y, 
        T * A, 
        T * x,

        int m,
        int n
     )
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;   
    // int c = blockIdx.x * blockDim.x + threadIdx.x;

    for(int k = 0; k < n; k++)
        {
            // *((T*)Acuda+ (r* n+c)) = *(A+ (r* n+c));
            // *((T*)xcuda+ (r* n+c)) = *(x+ (r* n+c));

            printf("%d",((T*)Acuda)[r * n + k]);
        }
}


template<typename T>
void multiplyMatrixGpuWrapper(
        void * Ycuda, 
        void * Acuda, 
        void * xcuda,

        T * Y, 
        T * A, 
        T * x,

        int m,
        int n,
        int width,
        T spacing
        ) 
{
    // dim3 block_dim(1,1,1);
    // dim3 grid_dim(1,1,1);

    std::cout<<"In the wrapper functions \n";
    int size = m*n*sizeof(T);
    std::cout<<*(A+1)<<std::endl;
    std::cout<<size<<std::endl;

    hipMalloc(&(Acuda),size);
    hipMemcpy(Acuda,A ,size, hipMemcpyHostToDevice);

    hipMalloc(&(xcuda),size); 
    hipMemcpy(xcuda,x , size,hipMemcpyHostToDevice);

    hipMalloc(&(Ycuda),size);
    hipMemcpy(Ycuda,Y ,size, hipMemcpyHostToDevice);

    // std::cout<<Acuda<<std::endl;

    multiplyGPU<<<3, 3>>>((T*)Ycuda, (T*)Acuda,(T*)xcuda, m,n ,width,spacing);
    hipDeviceSynchronize();

    hipMemcpy(Y,Ycuda ,size, hipMemcpyHostToDevice);

    hipFree(Ycuda);
    hipFree(xcuda);
    hipFree(Acuda);

}


template void multiplyMatrixGpuWrapper<int>
(
    void * Ycuda, void * Acuda, void * xcuda,
    int * Y, int * A, int * x,
    int m,int n,int width,int type
);
template void multiplyMatrixGpuWrapper<short>
(
    void * Ycuda, void * Acuda, void * xcuda,
    short * Y, short * A, short * x,
    int m,int n,int width,short type
);

template void multiplyMatrixGpuWrapper<long long>
(
    void * Ycuda, void * Acuda, void * xcuda,
    long long* Y, long long * A, long long* x,
    int m,int n,int width,long long type
);

