#include "hip/hip_runtime.h"
#include "cudaMul.cuh"
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template<typename T>
__global__  void multiplyGPU
    (
        T * Ycuda, 
        T * Acuda, 
        T * xcuda,
        int m,
        int n
    )
{
    int blkindex = blockIdx.x*blockDim.x;
    int blkindex_lookahead = blockIdx.x*blockDim.x+blockDim.x;
    int j=0;
    int i = blkindex;
    for( ; i < blkindex_lookahead ; i++)
    {
        Ycuda[threadIdx.x+blkindex] = Ycuda[threadIdx.x+blkindex] + (Acuda[i])*xcuda[(j+threadIdx.x+(blockDim.x-1)*j)];
        // printf("iterator value = i = %d threadIdx = %d ,value = %d , block = %d , A %d, x %d \n",i,threadIdx.x,value, blockIdx.x,*(Acuda+i),(j+threadIdx.x+(blockDim.x-1)*j));
        j++;
    }

}


template<typename T>
__global__  void GPUMemcpy
    (
        T * Acuda, 
        T * xcuda,

        T * Y,
        T * A, 
        T * x,
        int m,
        int n
    )
{
    //TODO @Explore more options 
}


template<typename T>
void multiplyMatrixGpuWrapper
        (
            void * Ycuda, 
            void * Acuda,
            void * xcuda,

            T * Y, 
            T * A, 
            T * x,

            int m,
            int n
        ) 
{
    int size = m*n*sizeof(T);


    hipMallocManaged(&Acuda, size);
    hipMallocManaged(&xcuda, size);
    hipMallocManaged(&Ycuda, size);

    for(int i=0; i< m*n;i++)
    {
        *((T*)xcuda+i)=*(x+i);
        *((T*)Acuda+i)=*(A+i);
    }

    // GPUMemcpy<<<m,n>>>((T*)Acuda,(T*)xcuda,Y,A,x, m,n);
    // hipDeviceSynchronize();
    multiplyGPU<<<m, n>>>((T*)Ycuda, (T*)Acuda,(T*)xcuda, m,n);
    hipDeviceSynchronize();

    for(int i=0; i< m*n;i++)
    {
        *(Y+i)=*((T*)Ycuda+i);
    }

    hipFree(Ycuda);
    hipFree(xcuda);
    hipFree(Acuda);

}


template void multiplyMatrixGpuWrapper<int>
(
    void * Ycuda, void * Acuda, void * xcuda,
    int * Y, int * A, int * x,
    int m,int n
);
template void multiplyMatrixGpuWrapper<short>
(
    void * Ycuda, void * Acuda, void * xcuda,
    short * Y, short * A, short * x,
    int m,int n
);

template void multiplyMatrixGpuWrapper<long long>
(
    void * Ycuda, void * Acuda, void * xcuda,
    long long* Y, long long * A, long long* x,
    int m,int n
);

